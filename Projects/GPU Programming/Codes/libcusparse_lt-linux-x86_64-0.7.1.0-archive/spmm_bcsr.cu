#include <hip/hip_runtime.h>
#include <cusparseLt.h>
#include <iostream>
#include <vector>

const char* cusparseLtGetErrorName(hipsparseStatus_t status) {
    switch (status) {
        case HIPSPARSE_STATUS_SUCCESS: return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED: return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED: return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE: return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH: return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR: return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED: return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR: return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default: return "CUSPARSE_STATUS_UNKNOWN";
    }
}

const char* cusparseLtGetErrorString(hipsparseStatus_t status) {
    return cusparseLtGetErrorName(status);  // For simplicity, same as name
}

#define CHECK_CUDA(func)                                                        \
{                                                                               \
    hipError_t status = (func);                                                \
    if (status != hipSuccess) {                                                \
        std::cerr << "CUDA error: " << hipGetErrorString(status) << std::endl;\
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUSPARSE(func)                                                    \
{                                                                               \
    hipsparseStatus_t status = (func);                                           \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                    \
        std::cerr << "cuSPARSELt error: " << cusparseLtGetErrorName(status)    \
                  << " - " << cusparseLtGetErrorString(status) << std::endl;   \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

int main() {
    // Dimensions
    int m = 64;  // rows of A and C
    int k = 64;  // cols of A and rows of B
    int n = 64;  // cols of B and C

    // Block sizes
    int block_size = 4;

    float alpha = 1.0f;
    float beta = 0.0f;

    // Example BSR block pattern (4 nonzero blocks)
    std::vector<int> bsr_row_ptr = {0, 2, 4};         // 2 block rows, each has 2 blocks
    std::vector<int> bsr_col_ind = {0, 2, 1, 3};      // Column indices for 4 blocks

    std::vector<float> bsr_val = {
        // Block (0,0)
          1,0,0,0,
          0,1,0,0,
          0,0,1,0,
          0,0,0,1,
          // Block (0,1)
          2,0,0,0,
          0,2,0,0,
          0,0,2,0,
          0,0,0,2,
        // Block (1,0)
          3,0,0,0,
          0,3,0,0,
          0,0,3,0,
          0,0,0,3,
        // Block (1,1)
          4,0,0,0,
          0,4,0,0,
          0,0,4,0,
          0,0,0,4
        };

    // Dense matrix B
    std::vector<float> hB(k * n, 1.0f); // B is k x n, initialized to ones

    // Allocate GPU memory and copy data from CPU to GPU
    float *dA, *dB, *dC;
    int *d_row_ptr, *d_col_ind;
    CHECK_CUDA(hipMalloc(&dA, sizeof(float) * bsr_val.size()));
    CHECK_CUDA(hipMalloc(&d_row_ptr, sizeof(int) * bsr_row_ptr.size()));
    CHECK_CUDA(hipMalloc(&d_col_ind, sizeof(int) * bsr_col_ind.size()));
    CHECK_CUDA(hipMalloc(&dB, sizeof(float) * k * n));
    CHECK_CUDA(hipMalloc(&dC, sizeof(float) * m * n));
    CHECK_CUDA(hipMemcpy(dA, bsr_val.data(), sizeof(float) * bsr_val.size(), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_row_ptr, bsr_row_ptr.data(), sizeof(int) * bsr_row_ptr.size(), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_col_ind, bsr_col_ind.data(), sizeof(int) * bsr_col_ind.size(), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, hB.data(), sizeof(float) * k * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(dC, 0, sizeof(float) * m * n));

    // cuSPARSELt handle
    cusparseLtHandle_t handle;
    CHECK_CUSPARSE(cusparseLtInit(&handle));

    // Matrix descriptors
    cusparseLtMatDescriptor_t matA, matB, matC;
    CHECK_CUSPARSE(cusparseLtStructuredDescriptorInit(&handle, &matA, m, k, m, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW, CUSPARSELT_SPARSITY_50_PERCENT));
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matB, k, n, k, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matC, m, n, m, 16, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    // Matmul descriptor
    cusparseLtMatmulDescriptor_t matmul;
    CHECK_CUSPARSE(cusparseLtMatmulDescriptorInit(&handle, &matmul,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &matA, &matB, &matC, &matC,
                                                  CUSPARSE_COMPUTE_32F));

    // Algorithm selection
    cusparseLtMatmulAlgSelection_t algSel;
    CHECK_CUSPARSE(cusparseLtMatmulAlgSelectionInit(&handle, &algSel, &matmul,
                                                    CUSPARSELT_MATMUL_ALG_DEFAULT));

    // Plan
    cusparseLtMatmulPlan_t plan;
    CHECK_CUSPARSE(cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &algSel));

    // Workspace
    size_t workspaceSize = 0;
    CHECK_CUSPARSE(cusparseLtMatmulGetWorkspace(&handle, &plan, &workspaceSize));

    void* dWorkspace = nullptr;
    if (workspaceSize > 0)
        CHECK_CUDA(hipMalloc(&dWorkspace, workspaceSize));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch SpMM
    CHECK_CUSPARSE(cusparseLtMatmul(&handle, &plan, &alpha, dA, dB, &beta, dC, dC, dWorkspace, nullptr, 0));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_ms = 0;
    hipEventElapsedTime(&elapsed_ms, start, stop);

    std::cout << "SpMM execution time: " << elapsed_ms << " ms" << std::endl;

    std::cout << "SpMM using cuSPARSELt completed successfully." << std::endl;

    // Allocate host memory and copy result from device
    float* h_C = (float*) malloc(m * n * sizeof(float));
    hipMemcpy(h_C, dC, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << h_C[i*n + j] << "\t";
        }
        std::cout << std::endl;
    }

    // Cleanup
    if (dWorkspace) hipFree(dWorkspace);
    hipFree(dA); hipFree(dB); hipFree(dC);

    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matA));
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matB));
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matC));
    CHECK_CUSPARSE(cusparseLtMatmulPlanDestroy(&plan));
    CHECK_CUSPARSE(cusparseLtDestroy(&handle));

    std::cout << "SpMM using cuSPARSELt completed successfully." << std::endl;

    free(h_C);

    return 0;
}
